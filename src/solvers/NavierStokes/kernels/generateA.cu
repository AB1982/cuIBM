#include "hip/hip_runtime.h"
/***************************************************************************//**
* \file generateA.cu
* \author Krishnan, A. (anush@bu.edu)
* \brief Contain definition of the kernels required to generate the matrix A
*/

#include <solvers/NavierStokes/kernels/generateA.h>

/**
* namespace kernels
* \brief Contain all the custom-written CUDA kernels
*/
namespace kernels
{

/**
* \brief Generate the matrix A
*
* \param ARows rows of the COO matrix A
* \param ACols columns of the COO matrix A
* \param AVals values of the COO matrix A
* \param MVals values of the COO matrix M
* \param LRows rows of the COO matrix L
* \param LCols columns of the COO matrix L
* \param LVals values of the COO matrix A
* \param ASize number of entries of the COO matrix A
* \param alpha implicit coefficient of the diffusive scheme
*/
__global__
void generateA(int *ARows, int *ACols, real *AVals, real *MVals, int *LRows, int *LCols, real *LVals, int ASize, real alpha)
{
	for (int I=threadIdx.x + blockIdx.x*blockDim.x; I<ASize; I += blockDim.x*gridDim.x)
	{
		ARows[I] = LRows[I];
		ACols[I] = LCols[I];
		AVals[I] = -alpha*LVals[I] + (LRows[I]==LCols[I])*MVals[LRows[I]];
	}
}

/**
* \brief Generate the matrix A for the direct forcing method
*
* \param ARows rows of the COO matrix A
* \param ACols columns of the COO matrix A
* \param AVals values of the COO matrix A
* \param MVals values of the COO matrix M
* \param LRows rows of the COO matrix L
* \param LCols columns of the COO matrix L
* \param LVals values of the COO matrix A
* \param ASize number of entries of the COO matrix A
* \param alpha implicit coefficient of the diffusive scheme
* \param tagsX
* \param tagsY
*/
__global__
void generateADirectForcing(int *ARows, int *ACols, real *AVals, real *MVals, int *LRows, int *LCols, real *LVals, int ASize, real alpha, int *tagsX, int *tagsY)
{
	for(int I=threadIdx.x + blockIdx.x*blockDim.x; I<ASize; I += blockDim.x*gridDim.x)
	{
		ARows[I] = LRows[I];
		ACols[I] = LCols[I];
		AVals[I] =   (tagsX[LRows[I]] == -1 && tagsY[LRows[I]] == -1)*(-alpha*LVals[I]) // if the current location is untagged, add -alpha*L
		           + (tagsX[LRows[I]] != -1 || tagsY[LRows[I]] != -1)*(-LVals[I]) // if the current location is tagged, add -L
		           + (LRows[I]==LCols[I])*MVals[LRows[I]]; // if it is a diagonal, add M
	}
}
	
} // end of namespace kernels
