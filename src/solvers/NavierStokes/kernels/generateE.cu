#include "hip/hip_runtime.h"
/***************************************************************************//**
* \file generateE.cu
* \author Krishnan, A. (anush@bu.edu)
* \brief Definition of the kernels required to generate elements of the interpolation matrix
*/

#include <solvers/NavierStokes/kernels/generateE.h>

/**
* \brief Discrete delta function defined by Roma et al. (1999)
*
* \param x x- or y- component of the vector defined between a body point and a grid point
* \param h grid-spacing near the body point
*
* \return the value of the discrete Delta function
*/
__device__ \
real dhRomaDeviceE(real x, real h)
{
	real r = fabs(x)/h;
	
	if(r>1.5)
		return 0.0;
	else if(r>0.5 && r<=1.5)
		return 1.0/(6*h)*( 5.0 - 3.0*r - sqrt(-3.0*(1-r)*(1-r) + 1.0) );
	else
		return 1.0/(3*h)*( 1.0 + sqrt(-3.0*r*r + 1.0) );
}

/**
* \brief Compute the discrete Delta function in 2D 
*		using the definition by Roma et al. (1999)
*
* \param x x-component of the vector defined between a body point and a grid point
* \param y y-component of the vector defined between a body point and a grid point
* \param h grid-spacing near the body point
*
* \return the discrete Delta function in 2D
*/
__device__ \
real deltaDeviceE(real x, real y, real h)
{
	return dhRomaDeviceE(x, h) * dhRomaDeviceE(y, h);
}

/**
* \namespace kernels
* \brief Contain all custom-written CUDA kernels
*/
namespace kernels
{
	
/**
* \brief Compute elements of the interpolation matrix on the host
*
* \param ERows rows of the COO matrix \c E
* \param ECols columns of the COO matrix \c E
* \param EVals values of the COO matrix \c E
* \param nx number of cells in the x-direction
* \param ny number of cells in the y-direction
* \param x x-component of the Eulerian grid points
* \param y y-component of the Eulerian grid points
* \param dx cell-widths in the x-direction
* \param totalPoints total number of body points (all bodies included)
* \param xB x-coordinate of the body points
* \param yB y-coordinate of the body points
* \param I x-index of the cell in which the body point is located
* \param J y-index of the cell in which the body point is located
*/
void generateEHost(int *ERows,  int *ECols,  real *EVals,
                   int nx, int ny, real *x, real *y, real *dx,
                   int totalPoints, real *xB, real *yB, int *I, int *J)
{
	for(int bodyIdx=0; bodyIdx<totalPoints; bodyIdx++)
	{
		int  Ib=I[bodyIdx],
		     Jb=J[bodyIdx],
		     EIdx  = bodyIdx*12,
		     i, j;

		real Dx = dx[Ib];
	
		// uB = integral (u * delta * dxdy)
		// E = E_hat * R^-1 => divide E_hat by Dx
	
		// populate x-components
		for(j=Jb-1; j<=Jb+1; j++)
		{
			for(i=Ib-2; i<=Ib+1; i++)
			{
				ERows[EIdx] = bodyIdx;
				ECols[EIdx] = j*(nx-1) + i;
				EVals[EIdx] = Dx*delta(x[i+1]-xB[bodyIdx], 0.5*(y[j]+y[j+1])-yB[bodyIdx], Dx);
				EIdx++;
			}
		}

		// populate y-components
		for(j=Jb-2; j<=Jb+1; j++)
		{
			for(i=Ib-1; i<=Ib+1; i++)
			{
				ERows[EIdx+12*totalPoints-12] = bodyIdx + totalPoints;
				ECols[EIdx+12*totalPoints-12] = j*nx + i + (nx-1)*ny;
				EVals[EIdx+12*totalPoints-12] = Dx*delta(0.5*(x[i]+x[i+1])-xB[bodyIdx], y[j+1]-yB[bodyIdx], Dx);
				EIdx++;
			}
		}
	}
}

/**
* \brief Kernel to compute an element of the interpolation matrix on the device
*
* \param ERows rows of the COO matrix \c E
* \param ECols columns of the COO matrix \c E
* \param EVals values of the COO matrix \c E
* \param nx number of cells in the x-direction
* \param ny number of cells in the y-direction
* \param x x-component of the Eulerian grid points
* \param y y-component of the Eulerian grid points
* \param dx cell-widths in the x-direction
* \param totalPoints total number of body points (all bodies included)
* \param xB x-coordinate of the body points
* \param yB y-coordinate of the body points
* \param I x-index of the cell in which the body point is located
* \param J y-index of the cell in which the body point is located
*/
__global__ \
void generateE(int *ERows,  int *ECols,  real *EVals,
               int nx, int ny, real *x, real *y, real *dx,
               int totalPoints, real *xB, real *yB, int *I, int *J)
{
	int bodyIdx = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(bodyIdx < totalPoints)
	{
		int  Ib=I[bodyIdx],
			 Jb=J[bodyIdx],
			 EIdx  = bodyIdx*12,
			 i, j;

		real Dx = dx[Ib];
	
		// uB = integral u * delta * dxdy = Ehat * u
		// E = Ehat * R^-1 => divide by Dx
		// E = Dx * delta
	
		// populate x-components
		for(j=Jb-1; j<=Jb+1; j++)
		{
			for(i=Ib-2; i<=Ib+1; i++)
			{
				ERows[EIdx] = bodyIdx;
				ECols[EIdx] = j*(nx-1) + i;
				EVals[EIdx] = Dx*deltaDeviceE(x[i+1]-xB[bodyIdx], 0.5*(y[j]+y[j+1])-yB[bodyIdx], Dx);
				EIdx++;
			}
		}

		// populate y-components
		for(j=Jb-2; j<=Jb+1; j++)
		{
			for(i=Ib-1; i<=Ib+1; i++)
			{
				ERows[EIdx+12*totalPoints-12] = bodyIdx + totalPoints;
				ECols[EIdx+12*totalPoints-12] = j*nx + i + (nx-1)*ny;
				EVals[EIdx+12*totalPoints-12] = Dx*deltaDeviceE(0.5*(x[i]+x[i+1])-xB[bodyIdx], y[j+1]-yB[bodyIdx], Dx);
				EIdx++;
			}
		}
	}
}

} // end of namespace kernels
