#include "hip/hip_runtime.h"
#include <solvers/NavierStokes/kernels/generateBC1.h>

namespace kernels
{

__global__
void bc1DirichletU(real *bc1, int N, int nx, int offset, int stride, real *dx, real C, real *bc)
{
	int idx	= threadIdx.x + blockIdx.x*blockDim.x;
	
	if ( idx >= N ) return;	/// check if idx too high
	
	int	I	= (offset + idx*stride),
		i	= I % (nx-1);
	bc1[I] += bc[idx] * C * 0.5*(dx[i] + dx[i+1]);
}

__global__
void bc1DirichletV(real *bc1, int N, int nx, int numU, int offset, int stride, real *dy, real C, real *bc, int numUbc)
{
	int idx	= threadIdx.x + blockIdx.x*blockDim.x;
		
	if ( idx >= N ) return;	/// check if idx too high
	
	int	I	= (offset + idx*stride),
		j	= I / nx;
	bc1[numU + I] += bc[idx+numUbc] * C * 0.5*(dy[j] + dy[j+1]);
}

__global__
void bc1ConvectiveU(real *bc1, int N, int nx, int offset, int stride, real *dx, real *dy, real C, real *bc, real *q, real alpha)
{
	int idx	= threadIdx.x + blockIdx.x*blockDim.x;
	
	if ( idx >= N ) return;	/// check if idx too high
	
	int	I = (offset + idx*stride),
		i = I % (nx-1),
		j = I / (nx-1);
	
	bc[idx] = (1.0-alpha)*bc[idx] + alpha*q[I]/dy[j];
	
	bc1[I] += bc[idx] * C * 0.5*(dx[i] + dx[i+1]);
}

__global__
void bc1ConvectiveV(real *bc1, int N, int nx, int numU, int offset, int stride, real *dx, real *dy, real C, real *bc, int numUbc, real *q, real alpha)
{
	int idx	= threadIdx.x + blockIdx.x*blockDim.x;
		
	if ( idx >= N ) return;	/// check if idx too high
	
	int	I = (offset + idx*stride),
		i = I % nx,
		j = I / nx;
	
	bc[idx+numUbc] = (1.0-alpha)*bc[idx+numUbc] + alpha*q[numU + I]/dx[i];
	
	bc1[numU + I] += bc[idx+numUbc] * C * 0.5*(dy[j] + dy[j+1]);
}

} // end of namespace kernels