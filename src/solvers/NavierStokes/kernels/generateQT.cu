#include "hip/hip_runtime.h"
/***************************************************************************//**
* \file generateQT.cu
* \author Krishnan, A. (anush@bu.edu)
* \brief Definition of the kernels required to generate the matrix \c QT
*/

#include <solvers/NavierStokes/kernels/generateQT.h>

/**
* \brief Calculate the discrete Delta function from Rome et al. (1999)
*
* \param x x- or y- component of the vector defined between two points
* \param h grid-spacing
*
* \return the value of the discrete delta function 
*/
__device__ \
real dhRomaDeviceQT(real x, real h)
{
	real r = fabs(x)/h;
	
	if(r>1.5)
		return 0.0;
	else if(r>0.5 && r<=1.5)
		return 1.0/(6*h)*( 5.0 - 3.0*r - sqrt(-3.0*(1-r)*(1-r) + 1.0) );
	else
		return 1.0/(3*h)*( 1.0 + sqrt(-3.0*r*r + 1.0) );
}

/**
* \brief Calculate the discrete delta function in 2D
*
* \param x x-component of the vector defined between two points
* \param y y-component of the vector defined between two points
* \param h grid-spacing
*
* \return the value of the discrete Delta function in 2D
*/
__device__ \
real deltaDeviceQT(real x, real y, real h)
{
	return dhRomaDeviceQT(x, h) * dhRomaDeviceQT(y, h);
}

/**
* \namespace kernels
* \brief Contain all custom-written CUDA kernels
*/
namespace kernels
{

/**
* \brief To be documented
*/
__global__ \
void updateQ(int *QRows, int *QCols, real *QVals, int QSize, int *tags)
{
	int I = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(I >= QSize) return;
	
	QVals[I] *= ( tags[QRows[I]] == -1 );
}

/**
* \brief To be documented
*/
__global__ \
void updateQ(int *QRows, int *QCols, real *QVals, int QSize, int *tagsX, int *tagsY)
{
	int I = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(I >= QSize) return;
	
	QVals[I] *= ( tagsX[QRows[I]] == -1 && tagsY[QRows[I]] == -1 );
}

/**
* \brief Generate the matrix \c QT.
*
* It actually generates the GT part (discrete divergence operator).
*
* \param QTRows row index of elements of the COO-format divergence matrix \c QT
* \param QTCols column index of elements of the COO-format divergence matrix \c QT
* \param QTVals value of elements of the COO-format divergence matrix \c QT
* \param nx number of cells in the x-direction
* \param ny number of cells in the y-direction
*/
void generateQT(int *QTRows, int *QTCols, real *QTVals, int nx, int ny)
{
	int  numU = (nx-1)*ny;
	
	int Iu, Iv;
	int row = 0;
	int num_elements = 0;
	
	/// QT is an (np + 2*nb) x nuv matrix
	
	/// Generate the GT part
	for(int j=0; j<ny; j++)
	{
		for(int i=0; i<nx; i++)
		{
			Iu = j*(nx-1) + i;
			Iv = j*nx + i + numU;
			
			if(i>0)
			{
				QTRows[num_elements] = row;
				QTCols[num_elements] = Iu - 1;
				QTVals[num_elements] = 1;
				num_elements++;
			}
			if(i<nx-1)
			{
				QTRows[num_elements] = row;
				QTCols[num_elements] = Iu;
				QTVals[num_elements] = -1;
				num_elements++;
			}
			if(j>0)
			{
				QTRows[num_elements] = row;
				QTCols[num_elements] = Iv - nx;
				QTVals[num_elements] = 1;
				num_elements++;
			}
			if(j<ny-1)
			{
				QTRows[num_elements] = row;
				QTCols[num_elements] = Iv;
				QTVals[num_elements] = -1;
				num_elements++;
			}
			row++;
		}
	}
}

/**
* \brief Kernel to update element of the matrix \c QT on the device.
*
* It also updates element of the interpolation matrix \c E.
*
* \param QTRows row index of elements of the COO-format matrix \c QT
* \param QTCols column index of elements of the COO-format matrix \c QT
* \param QTVals value of elements of the COO-format matrix \c QT
* \param ERows row index of elements of the COO-format interpolation matrix \c E
* \param ECols column index of elements of the COO-format interpolation matrix \c E
* \param EVals value of elements of the COO-format interpolation matrix \c E
* \param nx number of cells in the x-direction
* \param ny number of cells in the y-direction
* \param x x-component of grid points
* \param y y-component of grid points
* \param dx cell widths in the x-direction
* \param totalPoints number of body points (all bodies included)
* \param xB x-component of body points (all bodies included)
* \param yB y-component of body points (all bodies included)
* \param I x-index of grid cells in which body points are located
* \param J y-index of grid cells in which body points are located
*/
__global__ \
void updateQT(int *QTRows, int *QTCols, real *QTVals,
              int *ERows,  int *ECols,  real *EVals,
              int nx, int ny, real *x, real *y, real *dx,
              int totalPoints, real *xB, real *yB, int *I, int *J)
{
	int bodyIdx = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(bodyIdx >= totalPoints)
		return;
	
	int  Ib=I[bodyIdx],
	     Jb=J[bodyIdx],
	     QTIdx = 4*nx*ny - 2*(nx+ny) + bodyIdx*12,
	     EIdx  = bodyIdx*12,
	     i, j;

	real Dx = dx[Ib];
	
	// populate x-components
	for(j=Jb-1; j<=Jb+1; j++)
	{
		for(i=Ib-2; i<=Ib+1; i++)
		{
			QTRows[QTIdx] = bodyIdx + nx*ny;
			ERows[EIdx] = bodyIdx;
			
			QTCols[QTIdx] = j*(nx-1) + i;
			ECols[EIdx] = QTCols[QTIdx];
			
			QTVals[QTIdx] = Dx*deltaDeviceQT(x[i+1]-xB[bodyIdx], 0.5*(y[j]+y[j+1])-yB[bodyIdx], Dx);
			EVals[EIdx] = QTVals[QTIdx];
			
			QTIdx++;
			EIdx++;
		}
	}

	// populate y-components
	for(j=Jb-2; j<=Jb+1; j++)
	{
		for(i=Ib-1; i<=Ib+1; i++)
		{
			QTRows[QTIdx+12*totalPoints-12] = bodyIdx + nx*ny + totalPoints;
			ERows[EIdx+12*totalPoints-12] = bodyIdx + totalPoints;
			
			QTCols[QTIdx+12*totalPoints-12] = j*nx + i + (nx-1)*ny;
			ECols[EIdx+12*totalPoints-12] = QTCols[QTIdx+12*totalPoints-12];
			
			QTVals[QTIdx+12*totalPoints-12] = Dx*deltaDeviceQT(0.5*(x[i]+x[i+1])-xB[bodyIdx], y[j+1]-yB[bodyIdx], Dx);
			EVals[EIdx+12*totalPoints-12] = QTVals[QTIdx+12*totalPoints-12];
			
			QTIdx++;
			EIdx++;
		}
	}
}

/**
* \brief Update the matrix \c QT on the host.
*
* It also updates the interpolation matrix \c E.
*
* \param QTRows row index of elements of the COO-format matrix \c QT
* \param QTCols column index of elements of the COO-format matrix \c QT
* \param QTVals value of elements of the COO-format matrix \c QT
* \param ERows row index of elements of the COO-format interpolation matrix \c E
* \param ECols column index of elements of the COO-format interpolation matrix \c E
* \param EVals value of elements of the COO-format interpolation matrix \c E
* \param nx number of cells in the x-direction
* \param ny number of cells in the y-direction
* \param x x-component of grid points
* \param y y-component of grid points
* \param dx cell widths in the x-direction
* \param totalPoints number of body points (all bodies included)
* \param xB x-component of body points (all bodies included)
* \param yB y-component of body points (all bodies included)
* \param I x-index of grid cells in which body points are located
* \param J y-index of grid cells in which body points are located
*/
void updateQTHost(int *QTRows, int *QTCols, real *QTVals,
              int *ERows,  int *ECols,  real *EVals,
              int nx, int ny, real *x, real *y, real *dx,
              int totalPoints, real *xB, real *yB, int *I, int *J)
{
	for(int bodyIdx=0; bodyIdx<totalPoints; bodyIdx++)
	{
		int  Ib=I[bodyIdx],
		     Jb=J[bodyIdx],
		     QTIdx = 4*nx*ny - 2*(nx+ny) + bodyIdx*12,
		     EIdx  = bodyIdx*12,
		     i, j;

		real Dx = dx[Ib];
	
		// populate x-components
		for(j=Jb-1; j<=Jb+1; j++)
		{
			for(i=Ib-2; i<=Ib+1; i++)
			{
				QTRows[QTIdx] = bodyIdx + nx*ny;
				ERows[EIdx] = bodyIdx;
			
				QTCols[QTIdx] = j*(nx-1) + i;
				ECols[EIdx] = QTCols[QTIdx];
			
				QTVals[QTIdx] = Dx*delta(x[i+1]-xB[bodyIdx], 0.5*(y[j]+y[j+1])-yB[bodyIdx], Dx);
				EVals[EIdx] = QTVals[QTIdx];
			
				QTIdx++;
				EIdx++;
			}
		}

		// populate y-components
		for(j=Jb-2; j<=Jb+1; j++)
		{
			for(i=Ib-1; i<=Ib+1; i++)
			{
				QTRows[QTIdx+12*totalPoints-12] = bodyIdx + nx*ny + totalPoints;
				ERows[EIdx+12*totalPoints-12] = bodyIdx + totalPoints;
			
				QTCols[QTIdx+12*totalPoints-12] = j*nx + i + (nx-1)*ny;
				ECols[EIdx+12*totalPoints-12] = QTCols[QTIdx+12*totalPoints-12];
			
				QTVals[QTIdx+12*totalPoints-12] = Dx*delta(0.5*(x[i]+x[i+1])-xB[bodyIdx], y[j+1]-yB[bodyIdx], Dx);
				EVals[EIdx+12*totalPoints-12] = QTVals[QTIdx+12*totalPoints-12];
			
				QTIdx++;
				EIdx++;
			}
		}
	}
}

} // end of namespace kernels
