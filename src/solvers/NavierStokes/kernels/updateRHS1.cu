#include "hip/hip_runtime.h"
#include <solvers/NavierStokes/kernels/updateRHS1.h>

#define BSZ 16

namespace kernels
{

// 1-d interpolation
/*__global__
void updateRHS1(real *rhs1, int numUV, int *tags)
{
	int	I = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(I>=numUV)
		return;
	
	rhs1[I] = rhs1[I]*(tags[I]==-1);
}*/

// 2-d interpolation
__global__
void updateRHS1(real *rhs1, int numUV, int *tagsX, int *tagsY)
{
	int	I = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(I>=numUV)
		return;
	
	rhs1[I] = rhs1[I]*(tagsX[I]==-1 && tagsY[I]==-1);
}

__global__
void updateRHS1X(real *rhs1, int nx, int ny, real dt, real *dx, int *tagsX, int *tagsY, real *coeffsX, real *coeffsY, real *uvX, real *uvY)
{
	int	I = blockIdx.x*blockDim.x + threadIdx.x;
	int i = I % (nx-1);
	
	if( I < (nx-1)*ny )
	{
		rhs1[I] = (tagsX[I]==-1 && tagsY[I]==-1)*rhs1[I] + ( (tagsX[I]!=-1)*(1.0-coeffsX[I])*(1.0-coeffsY[I])*uvX[I] + (tagsY[I]!=-1)*coeffsX[I]*(1.0-coeffsY[I])*uvY[I] ) * 0.5*(dx[i+1]+dx[i]) / dt;
	}
}

__global__
void updateRHS1Y(real *rhs1, int nx, int ny, real dt, real *dy, int *tagsX, int *tagsY, real *coeffsX, real *coeffsY, real *uvX, real *uvY)
{
	int numU = (nx-1)*ny;
	int	I = blockIdx.x*blockDim.x + threadIdx.x + numU;
	int j = (I-numU) / nx;
	
	if( I < numU + nx*(ny-1) )
	{
		rhs1[I] = (tagsX[I]==-1 && tagsY[I]==-1)*rhs1[I] + ( (tagsX[I]!=-1)*(1.0-coeffsX[I])*(1.0-coeffsY[I])*uvX[I] + (tagsY[I]!=-1)*coeffsX[I]*(1.0-coeffsY[I])*uvY[I] ) * 0.5*(dy[j+1]+dy[j]) / dt;
	}
}

} // end of namespace kernels
