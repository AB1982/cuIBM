#include "hip/hip_runtime.h"
/*
*  Copyright (C) 2012 by Anush Krishnan, Simon Layton, Lorena Barba
*
*  Permission is hereby granted, free of charge, to any person obtaining a copy
*  of this software and associated documentation files (the "Software"), to deal
*  in the Software without restriction, including without limitation the rights
*  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
*  copies of the Software, and to permit persons to whom the Software is
*  furnished to do so, subject to the following conditions:
*
*  The above copyright notice and this permission notice shall be included in
*  all copies or substantial portions of the Software.
*
*  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
*  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
*  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
*  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
*  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
*  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
*  THE SOFTWARE.
*/

#include <solvers/NavierStokes/kernels/calculateForce.h>

namespace kernels
{

__global__
void dragLeftRight(real *FxY, real *q, real *lambda, real *dx, real *dy,
                   int nx, int ny, int I, int J, int ncx, int ncy)
{
	int  idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx >= ncy)
		return;
	int  Ip = (J+idx)*nx + I,
	     Iu = (J+idx)*(nx-1) + (I-1);
	FxY[idx] = -(
	              // multiply the pressure with the surface area to get p dy
	              (lambda[Ip+ncx]-lambda[Ip-1])*dy[J+idx]
	              +
	              // divide q^2 by dy, so that just u^2 dy is obtained
	              (
	                0.25*(q[Iu+ncx+1] + q[Iu+ncx])*(q[Iu+ncx+1] + q[Iu+ncx]) -
	                0.25*(q[Iu] + q[Iu-1])*(q[Iu] + q[Iu-1])
	              )/dy[J+idx]
	              -
	              // no multiplication or division since du/dx dy = dq/dx
	              (
	                2*(q[Iu+ncx+1] - q[Iu+ncx])/dx[I+ncx] -
	                2*(q[Iu] - q[Iu-1])/dx[I-1]
	              )
	            );	
}

__global__
void dragBottomTop(real *FxX, real *q, real *dx, real *dy,
                   int nx, int ny, int I, int J, int ncx, int ncy)
{
	int  idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx >= ncx)
		return;
	int  Iu = J*(nx-1) + (I+idx-1),
	     Iv = (nx-1)*ny + (J-1)*nx + I+idx;
	FxX[idx] = -(
	              // multiply by dS
	              (
	                0.25 * (q[Iu+ncy*(nx-1)]/dy[J+ncy] + q[Iu+(ncy-1)*(nx-1)]/dy[J+ncy-1])
	                     * (q[Iv+ncy*nx]/dx[I+idx] + q[Iv+ncy*nx-1]/dx[I+idx-1])
	                -
	                0.25 * (q[Iu]/dy[J] + q[Iu-(nx-1)]/dy[J-1]) * (q[Iv]/dx[I+idx] + q[Iv-1]/dx[I+idx-1])
	              )
	              -
	              // multiply by dS (cannot use the leftRight trick in this case)
	              (
	                (
	                  (q[Iu+ncy*(nx-1)]/dy[J+ncy] - q[Iu+(ncy-1)*(nx-1)]/dy[J+ncy-1])/2.0/(dy[J+ncy]+dy[J+ncy-1]) +
	                  (q[Iv+ncy*nx]/dx[I+idx] - q[Iv+ncy*nx-1]/dx[I+idx-1])/2.0/(dx[I+idx]+dx[I+idx-1])
	                ) -
	                (
	                  (q[Iu]/dy[J] - q[Iu-(nx-1)]/dy[J-1])/2.0/(dy[J]+dy[J-1]) +
	                  (q[Iv]/dx[I+idx] - q[Iv-1]/dx[I+idx-1])/2.0/(dx[I+idx]+dx[I+idx-1])
	                )
	              )
	            )*0.5*(dx[I+idx]+dx[I+idx-1]);	

}

__global__
void dragUnsteady(real *FxU, real *q, real *qOld, real *dx, real *dy, real dt,
                   int nx, int ny, int I, int J, int ncx, int ncy)
{
	int  idx = threadIdx.x + blockIdx.x*blockDim.x;
	     
	if(idx >= (ncx+1)*ncy)
		return;
	
	int i = idx%(ncx+1),
	    j = idx/(ncx+1);
	    
	int Iu = (J+j)*(nx-1) + (I+i-1);
	
	FxU[idx] = - ( q[Iu] - qOld[Iu])/dt * 0.5*(dx[I+i]+dx[I+i-1]);
}

} // end of namespace kernels
