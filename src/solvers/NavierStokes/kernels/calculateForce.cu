#include "hip/hip_runtime.h"
#include <solvers/NavierStokes/kernels/calculateForce.h>

#define BSZ 16

namespace kernels
{

__global__
void dragLeftRight(real *FxX, real *q, real *lambda, real nu, real *dx, real *dy,
                   int nx, int ny, int I, int J, int ncx, int ncy)
{
	int  idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx >= ncy)
		return;
	int  Ip = (J+idx)*nx + I,
	     Iu = (J+idx)*(nx-1) + (I-1);
	FxX[idx] = -(
	              // multiply the pressure with the surface area to get p dy
	              (lambda[Ip+ncx]-lambda[Ip-1])*dy[J+idx]
	              +
	              // divide q^2 by dy, so that just u^2 dy is obtained
	              (
	                  0.25*(q[Iu+ncx+1] + q[Iu+ncx])*(q[Iu+ncx+1] + q[Iu+ncx])
	                - 0.25*(q[Iu] + q[Iu-1])*(q[Iu] + q[Iu-1])
	              )/dy[J+idx]
	              -
	              // no multiplication or division since du/dx dy = dq/dx
	              nu*
	              (
	                  (q[Iu+ncx+1] - q[Iu+ncx])/dx[I+ncx]
	                - (q[Iu] - q[Iu-1])/dx[I-1]
	              )
	            );
}

__global__
void dragBottomTop(real *FxY, real *q, real nu, real *dx, real *dy,
                   int nx, int ny, int I, int J, int ncx, int ncy)
{
	int  idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx > ncx)
		return;
	int  Iu = J*(nx-1) + (I-1+idx),
	     Iv = (nx-1)*ny + (J-1)*nx + I+idx;
	FxY[idx] = -(
	              // multiply by dS
	              (
	                0.25 * ( q[Iu+ncy*(nx-1)]/dy[J+ncy] + q[Iu+(ncy-1)*(nx-1)]/dy[J+ncy-1] )
	                     * ( q[Iv+ncy*nx]/dx[I+idx] + q[Iv+ncy*nx-1]/dx[I+idx-1] )
	                -
	                0.25 * ( q[Iu]/dy[J] + q[Iu-(nx-1)]/dy[J-1] ) 
	                     * ( q[Iv]/dx[I+idx] + q[Iv-1]/dx[I+idx-1] )
	              )
	              -
	              // multiply by dS (cannot use the leftRight trick in this case)
	              nu*
	              (
	                (
	                  (q[Iu+ncy*(nx-1)]/dy[J+ncy] - q[Iu+(ncy-1)*(nx-1)]/dy[J+ncy-1])/2.0/(dy[J+ncy]+dy[J+ncy-1]) +
	                  (q[Iv+ncy*nx]/dx[I+idx] - q[Iv+ncy*nx-1]/dx[I+idx-1])/2.0/(dx[I+idx]+dx[I+idx-1])
	                ) 
	                -
	                (
	                  (q[Iu]/dy[J] - q[Iu-(nx-1)]/dy[J-1])/2.0/(dy[J]+dy[J-1]) +
	                  (q[Iv]/dx[I+idx] - q[Iv-1]/dx[I+idx-1])/2.0/(dx[I+idx]+dx[I+idx-1])
	                )
	              )
	            )*0.5*(dx[I+idx]+dx[I+idx-1]);

}

__global__
void dragUnsteady(real *FxU, real *q, real *qOld, real *dx, real *dy, real dt,
                   int nx, int ny, int I, int J, int ncx, int ncy)
{
	int  idx = threadIdx.x + blockIdx.x*blockDim.x;
	     
	if(idx >= (ncx+1)*ncy)
		return;
	
	int i = idx%(ncx+1),
	    j = idx/(ncx+1);
	    
	int Iu = (J+j)*(nx-1) + (I-1+i);
	
	FxU[idx] = - (q[Iu] - qOld[Iu])/dt * 0.5*(dx[I+i]+dx[I-1+i]);
}

__global__
void liftLeftRight(real *FyX, real *q, real nu, real *dx, real *dy,
                   int nx, int ny, int I, int J, int ncx, int ncy)
{
	int  idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx > ncy)
		return;
	int  Iu = (J+idx)*(nx-1) + (I-1),
	     Iv = (nx-1)*ny + (J-1+idx)*nx + I;
	FyX[idx] = -(
	              // multiply by dS
	              (
	                0.25 * ( q[Iu+ncx]/dy[J+idx] + q[Iu+ncx-(nx-1)]/dy[J-1+idx] )
	                     * ( q[Iv+ncx]/dx[I+ncx] + q[Iv+ncx-1]/dx[I+ncx-1] )
	                -
	                0.25 * ( q[Iu]/dy[J+idx] + q[Iu-(nx-1)]/dy[J-1+idx] )
	                     * ( q[Iv]/dx[I] + q[Iv-1]/dx[I-1] )
	              )
	              -
	              // multiply by dS (cannot use the leftRight trick in this case)
	              nu*
	              (
	                (
	                  (q[Iu+ncx]/dy[J+idx] - q[Iu+ncx-(nx-1)]/dy[J-1+idx])/2.0/(dy[J+idx]+dy[J-1+idx]) +
	                  (q[Iv+ncx]/dx[I+ncx] - q[Iv+ncx-1]/dx[I+ncx-1])/2.0/(dx[I+ncx]+dx[I+ncx-1])
	                ) 
	                -
	                (
	                  (q[Iu]/dy[J+idx] - q[Iu-(nx-1)]/dy[J-1+idx])/2.0/(dy[J+idx]+dy[J-1+idx]) +
	                  (q[Iv]/dx[I] - q[Iv-1]/dx[I-1])/2.0/(dx[I]+dx[I-1])
	                )
	              )
	            )*0.5*(dy[J+idx]+dy[J-1+idx]);
}

__global__
void liftBottomTop(real *FyY, real *q, real *lambda, real nu, real *dx, real *dy,
                   int nx, int ny, int I, int J, int ncx, int ncy)
{
	int  idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx >= ncx)
		return;
	int  Ip = J*nx + I+idx,
	     Iv = (nx-1)*ny + (J-1)*nx + I+idx;
	FyY[idx] = -(
	              // multiply the pressure with the surface area to get p dx
	              (lambda[Ip+ncy*nx]-lambda[Ip-nx])*dx[I+idx]
	              +
	              // divide q^2 by dx, so that just v^2 dx is obtained
	              (
	                  0.25*(q[Iv+(ncy+1)*nx] + q[Iv+ncy*nx])*(q[Iv+(ncy+1)*nx] + q[Iv+ncy*nx])
	                - 0.25*(q[Iv] + q[Iv-nx])*(q[Iv] + q[Iv-nx])
	              )/dx[I+idx]
	              -
	              // no multiplication or division since dv/dy dx = dq/dy
	              nu*
	              (
	                  (q[Iv+(ncy+1)*nx] - q[Iv+ncy*nx])/dy[J+ncy]
	                - (q[Iv] - q[Iv-nx])/dy[J-1]
	              )
	            );
}
            
__global__
void liftUnsteady(real *FyU, real *q, real *qOld, real *dx, real *dy, real dt,
                  int nx, int ny, int I, int J, int ncx, int ncy)
{
	int  idx = threadIdx.x + blockIdx.x*blockDim.x;
	     
	if( idx >= ncx*(ncy+1) )
		return;
	
	int i = idx%ncx,
	    j = idx/ncx;
	    
	int Iv = (J-1+j)*nx + (I+i) + (nx-1)*ny;

	FyU[idx] = - (q[Iv] - qOld[Iv])/dt * 0.5*(dy[J+j]+dy[J-1+j]);
}

__global__
void forceX(real *f, real *q, real *rn, int *tags,
            int nx, int ny, real *dx, real *dy,
            real dt, real alpha, real nu)
{
	int bx	= blockIdx.x,
		by	= blockIdx.y,
		i	= threadIdx.x,
		j	= threadIdx.y;
	
	// work out global index of first point in block
	int I = (BSZ-2)*bx + i,
	    J = (BSZ-2)*by + j;
	
	if (I >= nx-1 || J >= ny) {
		return;
	}

	int  Gidx_x = J*(nx-1) + I;

	real dTerm;
	
	__shared__ real u[BSZ][BSZ];
						
	__shared__ real Dx[BSZ][BSZ], Dy[BSZ][BSZ];
	
	Dy[j][i] = dy[J];
	Dx[j][i] = dx[I];
	
	/// transfer from global to shared memory
	u[j][i] = q[Gidx_x]/Dy[j][i];
	__syncthreads();
	
	/// check bounds for convective term in the x-direction
	int global_check = ( I==0 || I==(nx-2) || J==0 || J==(ny-1) ),		///< check if we compute globally
	    block_check  = ( i==0 || i==(BSZ-1) || j==0 || j==(BSZ-1) );	///< check if element within block computes
	
	/// X-component
	if( !(global_check || block_check) )
	{
		dTerm = alpha*nu*2.0*( \
						 ( Dx[j][i]*u[j][i+1] - (Dx[j][i]+Dx[j][i+1])*u[j][i] + Dx[j][i+1]*u[j][i-1] ) / ( Dx[j][i]*Dx[j][i+1]*(Dx[j][i]+Dx[j][i+1]) ) \
					   
					   + 4.0*( (Dy[j][i]+Dy[j-1][i])*u[j+1][i] - (Dy[j-1][i] + 2.0*Dy[j][i] + Dy[j+1][i])*u[j][i] + (Dy[j][i]+Dy[j+1][i])*u[j-1][i] ) \
							/( (Dy[j][i]+Dy[j-1][i]) * (Dy[j-1][i] + 2.0*Dy[j][i] + Dy[j+1][i]) * (Dy[j][i]+Dy[j+1][i]) ) \
					     );
		
		f[Gidx_x] = ( u[j][i]/dt - dTerm - rn[Gidx_x]/(0.5*(Dx[j][i]+Dx[j][i+1])) ) * (!(tags[Gidx_x]==-1));
	}
}

__global__
void forceY(){}

} // end of namespace kernels
