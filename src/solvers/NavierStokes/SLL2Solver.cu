#include "hip/hip_runtime.h"
/*
*  Copyright (C) 2012 by Anush Krishnan, Simon Layton, Lorena Barba
*
*  Permission is hereby granted, free of charge, to any person obtaining a copy
*  of this software and associated documentation files (the "Software"), to deal
*  in the Software without restriction, including without limitation the rights
*  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
*  copies of the Software, and to permit persons to whom the Software is
*  furnished to do so, subject to the following conditions:
*
*  The above copyright notice and this permission notice shall be included in
*  all copies or substantial portions of the Software.
*
*  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
*  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
*  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
*  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
*  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
*  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
*  THE SOFTWARE.
*/

#include <solvers/NavierStokes/SLL2Solver.h>
#include <sys/stat.h>
#include <cusp/io/matrix_market.h>

//##############################################################################
//                           LINEAR SOLVES
//##############################################################################

template <typename memoryType>
void SLL2Solver<memoryType>::projectionStep()
{
	NavierStokesSolver<memoryType>::logger.startTimer("projectionStep");
	
	// Calculate qTilde ========================================================
	
	cusp::multiply(NavierStokesSolver<memoryType>::Q, NavierStokesSolver<memoryType>::lambda, NavierStokesSolver<memoryType>::temp1);
	cusp::multiply(NavierStokesSolver<memoryType>::BN, NavierStokesSolver<memoryType>::temp1, SuLaiLinSolver<memoryType>::qTilde);
	cusp::blas::axpby(NavierStokesSolver<memoryType>::qStar, SuLaiLinSolver<memoryType>::qTilde, SuLaiLinSolver<memoryType>::qTilde, 1.0, -1.0);
	
	// Solve for f =============================================================
	
	SuLaiLinSolver<memoryType>::assembleRHS3();  // assemble rhs3 to solve for f
		
	//cusp::io::write_matrix_market_file(F, "F.mtx");
	cusp::io::write_matrix_market_file(SuLaiLinSolver<memoryType>::rhs3, "rhs3.mtx");
	
	int maxIters = 10000;
	int relTol = 1e-5;
	
	cusp::default_monitor<real> sys3Mon(SuLaiLinSolver<memoryType>::rhs3, maxIters, relTol);
	//cusp::krylov::cg(F, f, rhs3, sys3Mon, *PC3);
	cusp::krylov::bicgstab(SuLaiLinSolver<memoryType>::F, SuLaiLinSolver<memoryType>::f, SuLaiLinSolver<memoryType>::rhs3, sys3Mon);//, *PC3);
	int iterationCount3 = sys3Mon.iteration_count();
	if (!sys3Mon.converged())
	{
		std::cout << "ERROR: Solve for f failed at time step " << NavierStokesSolver<memoryType>::timeStep << std::endl;
		std::cout << "Iterations   : " << iterationCount3 << std::endl;          
		std::cout << "Residual norm: " << sys3Mon.residual_norm() << std::endl;
		std::cout << "Tolerance    : " << sys3Mon.tolerance() << std::endl;
		std::exit(-1);
	}
	
	// Obtain q^n+1 ===============================================================
	
	cusp::multiply(SuLaiLinSolver<memoryType>::ET, SuLaiLinSolver<memoryType>::f, SuLaiLinSolver<memoryType>::temp3);
	cusp::multiply(NavierStokesSolver<memoryType>::BN, SuLaiLinSolver<memoryType>::temp3, NavierStokesSolver<memoryType>::q);
	cusp::blas::axpby(SuLaiLinSolver<memoryType>::qTilde, NavierStokesSolver<memoryType>::q, NavierStokesSolver<memoryType>::q, 1.0, -1.0);

	NavierStokesSolver<memoryType>::logger.stopTimer("projectionStep");
	
}

template class SLL2Solver<host_memory>;
template class SLL2Solver<device_memory>;
