/***************************************************************************//**
 * \file io.cu
 * \brief Implementation of the functions of the namespace \c io.
 */


#include <sys/stat.h>

#include "io.h"
#include "utilities/types.h"
#include "utilities/boundaryCondition.h"


/**
 * \brief Converts a string to a number.
 *
 * \param str a string
 *
 * \return a number (\c real or \c integer)
 */
template <typename T>
T toNumber(std::string str)
{
  T num;
  std::stringstream ss(str); //turn the string into a stream
  ss >> num; //convert
  return num;
}


/**
 * \namespace io
 * \brief Contains functions related to I/O tasks.
 */
namespace io
{

/**
 * \brief Splits a string given a delimiter.
 *
 * \param s the string to split
 * \param delim the delimiter
 * \param elems the vector that contains the different elements of the string
 *
 * \return a vector that contains the different elements of the string
 */
std::vector<std::string> &split(const std::string &s, char delim, std::vector<std::string> &elems)
{
  std::stringstream ss(s);
  std::string item;
  while (std::getline(ss, item, delim)) {
      elems.push_back(item);
  }
  return elems;
}


/**
 * \brief Splits a string given a delimiter.
 *
 * \param s the string to split
 * \param delim the delimiter
 *
 * \return a vector that contains the different elements of the string
 */
std::vector<std::string> split(const std::string &s, char delim)
{
  std::vector<std::string> elems;
  split(s, delim, elems);
  return elems;
}


//##############################################################################
//                                 INPUT
//##############################################################################

/**
 * \brief Reads data inputs from the command-line and the simulation files.
 *
 * \param argc number of arguments in the command-line
 * \param argv command-line arguments
 * \param DB database that contains all the simulation parameters
 * \param D object of the class \c domain that contains the computational grid
 */
void readInputs(int argc, char **argv, parameterDB &DB, domain &D)
{
	// get a default database
	initialiseDefaultDB(DB);
	
	// first pass of command line arguments
	commandLineParse1(argc, argv, DB);
	
	// case folder
	std::string folder = DB["inputs"]["caseFolder"].get<std::string>();
	std::string fname;
	
	// read the simulation file
	fname = folder + "/simParams.yaml";
	parseSimulationFile(fname, DB);
	
	// read the flow file
	fname = folder + "/flow.yaml";
	parseFlowFile(fname, DB);

	// read the domain file
	fname = folder + "/domain.yaml";
	parseDomainFile(fname, D);
	
	// read the body file
	fname = folder + "/bodies.yaml";
	parseBodiesFile(fname, DB);
	
	// second pass of command line -- overwrite values in DB
	commandLineParse2(argc, argv, DB);
}


/**
 * \brief Initializes the database with default values.
 *
 * \param DB database that contains all the simulation parameters
 */
void initialiseDefaultDB(parameterDB &DB)
{
	DB["inputs"] = componentParameter();
	DB["flow"] = componentParameter();
	DB["simulation"] = componentParameter();
	DB["velocitySolve"] = componentParameter();
	DB["PoissonSolve"] = componentParameter();

	// default input files
	std::string inputs = "inputs";
	DB[inputs]["caseFolder"].set<std::string>("cases/cylinder/Re40");
	DB[inputs]["deviceNumber"].set<int>(0);

	// flow parameters
	std::string flow = "flow";
	DB[flow]["nu"].set<real>(0.01);
	DB[flow]["uInitial"].set<real>(1.0);
	DB[flow]["vInitial"].set<real>(0.0);
	DB[flow]["numBodies"].set<int>(0);
	std::vector<body> *bodyVec = new std::vector<body>;
	DB[flow]["bodies"].set<std::vector<body> *>(bodyVec);

	// boundary conditions
	boundaryCondition **bc = new boundaryCondition*[4];
	for (int i=0; i<4; i++)
		bc[i] = new boundaryCondition[2];
	DB[flow]["boundaryConditions"].set<boundaryCondition **>(bc);

	// simulation parameters
	std::string sim = "simulation";
	DB[sim]["dt"].set<real>(0.02);
	DB[sim]["nt"].set<int>(100);
	DB[sim]["nsave"].set<int>(100);
	DB[sim]["startStep"].set<bool>(0);
	DB[sim]["convTimeScheme"].set<timeScheme>(EULER_EXPLICIT);
	DB[sim]["diffTimeScheme"].set<timeScheme>(EULER_IMPLICIT);
	DB[sim]["ibmScheme"].set<ibmScheme>(TAIRA_COLONIUS);
	DB[sim]["interpolationType"].set<interpolationType>(LINEAR);

	// velocity solver
	std::string solver = "velocitySolve";
	DB[solver]["solver"].set<std::string>("CG");
	DB[solver]["preconditioner"].set<preconditionerType>(DIAGONAL);
	DB[solver]["rTol"].set<real>(1.0E-05);
	DB[solver]["aTol"].set<real>(1.0E-50);
	DB[solver]["maxIterations"].set<int>(10000);

	// Poisson solver
	solver = "PoissonSolve";
	DB[solver]["solver"].set<std::string>("CG");
	DB[solver]["preconditioner"].set<preconditionerType>(DIAGONAL);
	DB[solver]["rTol"].set<real>(1.0E-05);
	DB[solver]["aTol"].set<real>(1.0E-50);
	DB[solver]["maxIterations"].set<int>(20000);
}


/**
 * \brief Parses the command-line to get the case folder name 
 *        and the device number.
 *
 * \param argc number of arguments in the command-line
 * \param argv arguments of the command-line
 * \param DB database that contains all the simulation parameters
 */
void commandLineParse1(int argc, char **argv, parameterDB &DB)
{
	for (int i=1; i<argc; i++)
	{
		if (strcmp(argv[i],"-caseFolder")==0)
		{
			i++;
			DB["inputs"]["caseFolder"].set<std::string>(std::string(argv[i]));
		}
		else if (strcmp(argv[i],"-deviceNumber")==0)
		{
			i++;
			int devNum = toNumber<int>(std::string(argv[i]));
			DB["inputs"]["deviceNumber"].set<int>(devNum);
			// sets devNum as the current device for the calling host thread
			hipSetDevice(devNum);
		}
	}
}

/**
 * \brief Overwrites parameters with additional arguments of the command-line. 
 *
 * \param argc number of arguments in the command-line
 * \param argv arguments of the command-line
 * \param DB database that contains all the simulation parameters
 */
void commandLineParse2(int argc, char **argv, parameterDB &DB)
{
	for (int i=1; i<argc; i++)
	{
		// kinematic viscosity
		if ( strcmp(argv[i],"-nu")==0 )
		{
			i++;
			DB["flow"]["nu"].set<real>(toNumber<real>(std::string(argv[i])));
		}
		// perturbation in the x-velocity
		if ( strcmp(argv[i],"-uPerturb")==0 )
		{
			i++;
			DB["flow"]["uPerturb"].set<real>(toNumber<real>(std::string(argv[i])));
		}
		// perturbation in the y-velocity
		if ( strcmp(argv[i],"-vPerturb")==0 )
		{
			i++;
			DB["flow"]["vPerturb"].set<real>(toNumber<real>(std::string(argv[i])));
		}
		// scale the CV with respect to the body
		if ( strcmp(argv[i],"-scaleCV")==0 )
		{
			i++;
			DB["simulation"]["scaleCV"].set<real>(toNumber<real>(std::string(argv[i])));
		}
		// frequency of saving the data
		if ( strcmp(argv[i],"-nsave")==0 )
		{
			i++;
			DB["simulation"]["nsave"].set<int>(toNumber<int>(std::string(argv[i])));
		}
		// total number of time steps
		if ( strcmp(argv[i],"-nt")==0 )
		{
			i++;
			DB["simulation"]["nt"].set<int>(toNumber<int>(std::string(argv[i])));
		}
		// size of time increment
		if ( strcmp(argv[i],"-dt")==0 )
		{
			i++;
			DB["simulation"]["dt"].set<real>(toNumber<real>(std::string(argv[i])));
		}
		// relative tolerance for the velocity solve
		if ( strcmp(argv[i],"-velocity-rtol")==0 )
		{
			i++;
			DB["velocitySolve"]["rTol"].set<real>(toNumber<real>(std::string(argv[i])));
		}
		// absolute tolerance for the velocity solve
		if ( strcmp(argv[i],"-velocity-atol")==0 )
		{
			i++;
			DB["velocitySolve"]["aTol"].set<real>(toNumber<real>(std::string(argv[i])));
		}
		// relative tolerance for the Poisson solve
		if ( strcmp(argv[i],"-poisson-rtol")==0 )
		{
			i++;
			DB["PoissonSolve"]["rTol"].set<real>(toNumber<real>(std::string(argv[i])));
		}
		// absolute tolerance for the Poisson solve
		if ( strcmp(argv[i],"-poisson-atol")==0 )
		{
			i++;
			DB["PoissonSolve"]["aTol"].set<real>(toNumber<real>(std::string(argv[i])));
		}
		// IBM Scheme
		if ( strcmp(argv[i],"-ibmScheme")==0 )
		{
			i++;
			if ( strcmp(argv[i],"NavierStokes")==0 )
				DB["simulation"]["ibmScheme"].set<ibmScheme>(NAVIER_STOKES);
			else
			if ( strcmp(argv[i],"TairaColonius")==0 )
				DB["simulation"]["ibmScheme"].set<ibmScheme>(TAIRA_COLONIUS);
			else
			if ( strcmp(argv[i],"DirectForcing")==0 )
				DB["simulation"]["ibmScheme"].set<ibmScheme>(DIRECT_FORCING);
			else
			if ( strcmp(argv[i],"FadlunEtAl")==0 )
				DB["simulation"]["ibmScheme"].set<ibmScheme>(FADLUN_ET_AL);
			else
			if ( strcmp(argv[i],"Diffusion")==0 )
				DB["simulation"]["ibmScheme"].set<ibmScheme>(DIFFUSION);
			else
			if ( strcmp(argv[i],"DFModified")==0 )
				DB["simulation"]["ibmScheme"].set<ibmScheme>(DF_MODIFIED);
			else
			if ( strcmp(argv[i],"FEAModified")==0 )
				DB["simulation"]["ibmScheme"].set<ibmScheme>(FEA_MODIFIED);
			else 
			if ( strcmp(argv[i],"DFImproved")==0 )
				DB["simulation"]["ibmScheme"].set<ibmScheme>(DF_IMPROVED);
		}
		// interpolation type for Eulerian direct forcing methods
		if ( strcmp(argv[i],"-interpolationType")==0 )
		{
			i++;
			if ( strcmp(argv[i],"constant")==0 )
				DB["simulation"]["interpolationType"].set<interpolationType>(CONSTANT);
			else
			if ( strcmp(argv[i],"linear")==0 )
				DB["simulation"]["interpolationType"].set<interpolationType>(LINEAR);
			else
			if ( strcmp(argv[i],"quadratic")==0 )
				DB["simulation"]["interpolationType"].set<interpolationType>(QUADRATIC);
		}
	}
}


//##############################################################################
//                                OUTPUT
//##############################################################################

/**
 * \brief Converts a \c preconditionerType to a \c std::string.
 *
 * \param s a preconditioner
 *
 * \return a string
 */
std::string stringFromPreconditionerType(preconditionerType s)
{
  if (s == NONE)
    return "None";
  else if (s == DIAGONAL)
    return "Diagonal";
  else if (s == SMOOTHED_AGGREGATION)
    return "Smoothed Aggregation";
  else if (s == AINV)
    return "Approximate Inverse";
  else
  {
  	printf("Error: Unknown preconditionerType.\n");
  	exit(-1);
  }
}


/**
 * \brief Converts a \c timeScheme to a \c std::string.
 *
 * \param s a time-integration scheme
 *
 * \return a string
 */
std::string stringFromTimeScheme(timeScheme s)
{
	if (s == EULER_EXPLICIT)
		return "Explicit Euler Method";
	else if (s == EULER_IMPLICIT)
		return "Implicit Euler Method";
	else if (s == ADAMS_BASHFORTH_2)
		return "2nd Order Adams-Bashforth";
	else if (s == CRANK_NICOLSON)
		return "Crank-Nicolson";
	else
	{
		printf("Error: Unknown timeScheme!\n");
		exit(-1);
	}
}


/**
 * \brief Prints the parameters of the simulation.
 *
 * \param DB database that contains all the simulation parameters
 * \param D information about the computational grid
 */
void printSimulationInfo(parameterDB &DB, domain &D)
{
	real dt = DB["simulation"]["dt"].get<real>(),
	     scaleCV = DB["simulation"]["scaleCV"].get<real>();
	int  nt = DB["simulation"]["nt"].get<int>(),
	     nsave = DB["simulation"]["nsave"].get<int>(),
	     startStep = DB["simulation"]["startStep"].get<int>();
	interpolationType interpType = DB["simulation"]["interpolationType"].get<interpolationType>();
	ibmScheme ibmSchm = DB["simulation"]["ibmScheme"].get<ibmScheme>();


  std::cout << '\n';
	
	std::cout << "\nFlow parameters" << '\n';
	std::cout << "---------------" << '\n';
	std::cout << "nu = " << DB["flow"]["nu"].get<real>() << '\n';

	std::cout << "\nDomain" << '\n';
	std::cout << "------" << '\n';
	std::cout << D.nx << " x " << D.ny << '\n';
	
	std::cout << "\nSimulation parameters" << '\n';
	std::cout << "---------------------" << '\n';
	std::cout << "dt = " << dt << '\n';
	std::cout << "scaleCV = " << scaleCV << '\n';
	std::cout << "startStep = " << startStep << '\n';
	std::cout << "nt = "    << nt << '\n';
	std::cout << "nsave = " << nsave << '\n';
	std::cout << "Convection time scheme = " << stringFromTimeScheme(DB["simulation"]["convTimeScheme"].get<timeScheme>()) << '\n';
	std::cout << "Diffusion time scheme  = " << stringFromTimeScheme(DB["simulation"]["diffTimeScheme"].get<timeScheme>()) << '\n';
	if (ibmSchm == FADLUN_ET_AL ||
			ibmSchm == DIRECT_FORCING ||
			ibmSchm == DIFFUSION ||
			ibmSchm == DF_IMPROVED ||
			ibmSchm == DF_MODIFIED ||
			ibmSchm == FEA_MODIFIED)
	{
		std::cout << "Interpolation type: ";
		switch(interpType)
		{
			case CONSTANT : std::cout << "Constant\n"; break;
			case LINEAR   : std::cout << "Linear\n"; break;
			case QUADRATIC: std::cout << "Quadratic\n"; break;
			default : std::cout << "Unknown\n"; break;
		}
	}
	
	std::cout << "\nVelocity Solve" << '\n';
	std::cout << "--------------" << '\n';
	std::cout << "Solver = " << DB["velocitySolve"]["solver"].get<std::string>() << '\n';
	std::cout << "Preconditioner = " << stringFromPreconditionerType(DB["velocitySolve"]["preconditioner"].get<preconditionerType>()) << '\n';
	std::cout << "Relative tolerance = " << DB["velocitySolve"]["rTol"].get<real>() << '\n';
	std::cout << "Absolute tolerance = " << DB["velocitySolve"]["aTol"].get<real>() << '\n';
	
	std::cout << "\nPoisson Solve" << '\n';
	std::cout << "-------------" << '\n';
	std::cout << "Solver = " << DB["PoissonSolve"]["solver"].get<std::string>() << '\n';
	std::cout << "Preconditioner = " << stringFromPreconditionerType(DB["PoissonSolve"]["preconditioner"].get<preconditionerType>()) << '\n';
	std::cout << "Relative tolerance = " << DB["PoissonSolve"]["rTol"].get<real>() << '\n';
	std::cout << "Absolute tolerance = " << DB["PoissonSolve"]["aTol"].get<real>() << '\n';
	
	std::cout << "\nOutput parameters" << '\n';
	std::cout << "-----------------" << '\n';
	std::cout << "Output folder = " << DB["inputs"]["caseFolder"].get<std::string>() << '\n';
	std::cout << "nsave = " << DB["simulation"]["nsave"].get<int>() << '\n';
	
	hipDeviceProp_t deviceProp;
	int gpu = DB["inputs"]["deviceNumber"].get<int>();
	hipGetDeviceProperties(&deviceProp, gpu);
	std::cout << "\nDevice Properties" << '\n';
	std::cout << "-----------------" << '\n';
	std::cout << "Name = " << deviceProp.name << '\n';
	std::cout << "Number = " << gpu << '\n';
	std::string ecc = deviceProp.ECCEnabled ? "yes" : "no";
	std::cout << "Compute capability = " << deviceProp.major << "." << deviceProp.minor << '\n';
	std::cout << "ECC Enabled = " << ecc << std::endl;
}


/**
 * \brief Prints the time spent to execute tasks.
 *
 * \param logger object that contains the name and time spent of tasks
 */
void printTimingInfo(Logger &logger)
{
	logger.printAllTime();
	std::cout << std::endl;
}


/**
 * \brief Writes information about the run into the file \a run.info.
 *
 * \param DB database that contains all the simulation parameters
 * \param D information about the computational grid
 */
void writeInfoFile(parameterDB &DB, domain &D)
{
	std::string folder = DB["inputs"]["caseFolder"].get<std::string>();
	std::ofstream infofile((folder+"/run.info").c_str());
	infofile << std::setw(20) << "--nx"  << "\t" << D.nx << '\n';
	infofile << std::setw(20) << "--ny"  << "\t" << D.ny << '\n';
	infofile << std::setw(20) << "--startStep" << "\t" << DB["simulation"]["startStep"].get<int>() << '\n';
	infofile << std::setw(20) << "--nt"     << "\t" << DB["simulation"]["nt"].get<int>() << '\n';
	infofile << std::setw(20) << "--nsave"  << "\t" << DB["simulation"]["nsave"].get<int>() << '\n';
	infofile << std::setw(20) << "--dt"     << "\t" << DB["simulation"]["dt"].get<real>() << '\n';
	infofile << std::setw(20) << "--vortlim"<< "\t" << 15 << '\n';
	infofile << std::setw(20) << "--folder" << "\t" << folder << '\n';
	infofile << std::setw(20) << "--nu"     << "\t" << DB["flow"]["nu"].get<real>() << '\n';
	infofile.close();
}


/**
 * \brief Writes grid-points coordinates into the file \a grid.
 *
 * \param caseFolder the directory of the simulation
 * \param D information about the computational grid
 */
void writeGrid(std::string &caseFolder, domain &D)
{
	std::stringstream out;
	out << caseFolder << "/grid";
	std::ofstream file(out.str().c_str(), std::ios::binary);
	file.write((char*)(&D.nx), sizeof(int));
	file.write((char*)(&D.x[0]), (D.nx+1)*sizeof(real));
	file.write((char*)(&D.ny), sizeof(int));
	file.write((char*)(&D.y[0]), (D.ny+1)*sizeof(real));
	file.close();
}


/**
 * \brief Writes numerical data at a given time-step (on the host).
 *
 * It creates a directory whose name is the time-step number
 * and writes the flux, the pressure (and eventually the body forces)
 * into the files \a q, \a lambda, respectively.
 *
 * \param caseFolder directory of the simulation
 * \param n the time-step number
 * \param q array that contains the fluxes
 * \param lambda array that contains the pressures (and eventually the body forces)
 * \param D information about the computational grid
 */
template <>
void writeData<vecH>(std::string &caseFolder, int n, vecH &q, vecH &lambda, domain &D)//, bodies &B)
{
	std::string path;
	std::stringstream out;
	int N;

	out << caseFolder << '/' << std::setfill('0') << std::setw(7) << n;
	path = out.str();

	mkdir(path.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);

	out.str("");
	out << path << "/q";
	std::ofstream file(out.str().c_str(), std::ios::binary);
	N = q.size();
	file.write((char*)(&N), sizeof(int));
	file.write((char*)(&q[0]), N*sizeof(real));
	file.close();

	out.str("");
	out << path << "/lambda";
	file.open(out.str().c_str(), std::ios::binary);
	N = lambda.size();
	file.write((char*)(&N), sizeof(int));
	file.write((char*)(&lambda[0]), N*sizeof(real));
	file.close();
	
	std::cout << "Data saved to folder " << path << std::endl;
}


/**
 * \brief Writes numerical data at a given time-step (on the device).
 *
 * It creates a directory whose name is the time-step number
 * and writes the flux, the pressure (and eventually the body forces)
 * into the files \a q, \a lambda, respectively.
 *
 * \param caseFolder directory of the simulation
 * \param n the time-step number
 * \param q array that contains the fluxes
 * \param lambda array that contains the pressures (and eventually the body forces)
 * \param D information about the computational grid
 */
template <>
void writeData<vecD>(std::string &caseFolder, int n, vecD &q, vecD &lambda, domain &D)//, bodies &B)
{
	vecH qH = q,
	     lambdaH = lambda;
	     
	writeData(caseFolder, n, qH, lambdaH, D);
}


/**
 * \brief Reads numerical data at a given time-step.
 *
 * \param caseFolder directory of the simulation
 * \param timeStep the time-step number
 * \param x array that to fill
 * \param name name of the file containing the variable
 */
void readData(std::string &caseFolder, int timeStep, real *x, std::string name)
{
	std::stringstream in;
	std::string inFilePath;
	int n;

	in << caseFolder << "/" << std::setfill('0') << std::setw(7) << timeStep << "/" << name;
	inFilePath = in.str();
	std::cout << "Reading fluxes from " << inFilePath << " ... ";
	std::ifstream inFile(inFilePath.c_str(), std::ifstream::binary);
	inFile.read((char*)(&n), sizeof(int));
	inFile.read((char*)(&x[0]), n*sizeof(real));
	inFile.close();
	std::cout << "done" << std::endl;
}


/**
 * \brief Prints device memory usage.
 *
 * \param label the label of the device
 */
void printDeviceMemoryUsage(char *label)
{
	size_t _free, _total;
	hipMemGetInfo(&_free, &_total);
	std::cout << '\n' << label << ": Memory Usage " << std::setprecision(3) << (_total-_free)/(1024.0*1024*1024) \
	          << " / " << std::setprecision(3) << _total/(1024.0*1024*1024) << " GB" << std::setprecision(6) << '\n' << std::endl;
}

} // end namespace io
