#include "hip/hip_runtime.h"
/**
 * \file parseDomainFile.cu
 * \brief Parse the input file \a domain.yaml to obtain information about the 
 *        computational grid.
 */


#include <fstream>

#include "io.h"
#include "yaml-cpp/yaml.h"


/**
 * \namespace io
 * \brief Contains functions related to I/O tasks.
 */
namespace io
{

/**
 * \brief Gets information from the parsed domain file.
 *
 * \param node the parsed file
 * \param D instance of the class \c domain to be filled
 */
void parseDomain(const YAML::Node &node, domain &D)
{
	std::string dir;
	real start, end, stretchRatio, h;
	int  numCells;
	
	dir = node["direction"].as<std::string>();
	start = node["start"].as<real>();
	
	if (dir=="x")
		D.nx = 0;
	else if(dir=="y")
		D.ny = 0;
	
	const YAML::Node &subDomains = node["subDomains"];
	// first pass
	for (unsigned int i=0; i<subDomains.size(); i++)
	{
		numCells = subDomains[i]["cells"].as<int>();
		if (dir=="x")
			D.nx += numCells;
		else if(dir=="y")
			D.ny += numCells;
	}
	
	// allocate memory
	int beg = 0;
	if(dir=="x")
	{
		D.x.resize(D.nx+1);
		D.dx.resize(D.nx);
		D.xD.resize(D.nx+1);
		D.dxD.resize(D.nx);
		D.x[beg] = start;
	}
	if(dir=="y")
	{
		D.y.resize(D.ny+1);
		D.dy.resize(D.ny);  
		D.yD.resize(D.ny+1);
		D.dyD.resize(D.ny);
		D.y[beg] = start;
	}
	
	// second pass
	for (unsigned int i=0; i<subDomains.size(); i++)
	{
		end = subDomains[i]["end"].as<real>();
		numCells = subDomains[i]["cells"].as<int>();
		stretchRatio = subDomains[i]["stretchRatio"].as<real>();
		
		if(fabs(stretchRatio-1.0) < 1.0e-6)
		{
			h = (end - start)/numCells;
			for(int j=beg; j<beg+numCells; j++)
			{
				if(dir=="x")
				{
					D.dx[j]  = h;
					D.x[j+1] = D.x[j] + D.dx[j];
				}
				else if(dir=="y")
				{
					D.dy[j]  = h;
					D.y[j+1] = D.y[j] + D.dy[j];
				} 
			}
		}
		else
		{
			h = (end - start)*(stretchRatio-1)/(pow(stretchRatio, numCells)-1);
			for(int j=beg; j<beg+numCells; j++)
			{
				if(dir=="x")
				{
					D.dx[j]  = h*pow(stretchRatio, j-beg);
					D.x[j+1] = D.x[j] + D.dx[j];
				}
				else if(dir=="y")
				{
					D.dy[j]  = h*pow(stretchRatio, j-beg);
					D.y[j+1] = D.y[j] + D.dy[j];
				}
			}
		}
		beg += numCells;
		start = end;
	}
	
	if(dir=="x")
	{
		D.xD  = D.x;
		D.dxD = D.dx;
	}
	else if(dir=="y")
	{
		D.yD  = D.y;
		D.dyD = D.dy;
	}
} // parseDomain


/**
 * \brief Parses the \a domain file and generates the computational grid.
 *
 * \param domFile the file that contains information about the computational grid
 * \param D instance of the class \c domain that will be filled with information about the computational grid
 */
void parseDomainFile(std::string &domFile, domain &D)
{
	YAML::Node nodes = YAML::LoadFile(domFile);
	for (unsigned int i=0; i<nodes.size(); i++)
		parseDomain(nodes[i], D);
		
	D.xu.resize(D.nx-1);
	D.yu.resize(D.ny);
	D.xv.resize(D.nx);
	D.yv.resize(D.ny-1);
	
	int i, j;
	for(i=0; i<D.nx-1; i++)
	{
		D.xu[i] = D.x[i+1];
		D.xv[i] = (D.x[i]+D.x[i+1])/2.0;
	}
	D.xv[i] = (D.x[i]+D.x[i+1])/2.0;
	
	for(j=0; j<D.ny-1; j++)
	{
		D.yu[j] = (D.y[j]+D.y[j+1])/2.0;
		D.yv[j] = D.y[j+1];
	}
	D.yu[j] = (D.y[j]+D.y[j+1])/2.0;
} // parseDomainFile

} // End of namespace io
